#include "hip/hip_runtime.h"
#ifndef __MAIN_CU
#define __MAIN_CU

#include <iostream>
#include "main.h"
#include "assert.h"

#define THREAD 1024

struct cuda_timer_t {
  float time;

  cuda_timer_t() {
    hipEventCreate(&start_);
    hipEventCreate(&stop_);
    hipEventRecord(start_);
  }

  ~cuda_timer_t() {
    hipEventDestroy(start_);
    hipEventDestroy(stop_);
  }

  void start() { hipEventRecord(start_); }
  
  float stop() {
    hipEventRecord(stop_);
    hipEventSynchronize(stop_);
    hipEventElapsedTime(&time, start_, stop_);

    return microseconds();
  }
  
  float microseconds() { return (long long)(1000 * time); }

 private:
  hipEvent_t start_, stop_;
};

void loadGraph(char * node_filename, char * edge_filename, Graph* d_graph) {

  // -------------------------
  // Read nodes

  IntT num_nodes, node_line_length;
  FILE * nodeFile = fopen(node_filename, "r");
  if (! nodeFile) {printf("Cannot open node file %s\n", node_filename); exit(1);}

  fscanf(nodeFile, "%lu %lu", & num_nodes, & node_line_length);
  IntT node_feat_dim = node_line_length - 1;

  FloatT * node_feats = (FloatT *) malloc(num_nodes * node_feat_dim * sizeof(FloatT));
  IntT * node_ids = (IntT *) malloc(num_nodes * sizeof(IntT));

  for (IntT node_idx = 0; node_idx < num_nodes; node_idx++) {
    fscanf(nodeFile, "%lu", node_ids + node_idx);
    assert(node_ids[node_idx] == node_idx);
    for (IntT feat_idx = 0; feat_idx < node_feat_dim; feat_idx ++) {
        fscanf(nodeFile, "%lf", (FloatT *) (node_feats + (node_feat_dim * node_idx) + feat_idx));
    }
  }

  // -------------------------
  // Read edges

  IntT num_edges, edge_line_length;
  FILE * edgeFile = fopen(edge_filename, "r");
  if (! edgeFile) {printf("Cannot open file %s\n", edge_filename); exit(1);}

  fscanf(edgeFile, "%lu %lu", & num_edges, & edge_line_length);
  IntT edge_feat_dim = edge_line_length - 2;

  FloatT * edge_feats = (FloatT *) malloc(num_edges * edge_feat_dim * sizeof(FloatT));
  IntT * srcs         = (IntT *) malloc(num_edges * sizeof(IntT));
  IntT * dsts         = (IntT *) malloc(num_edges * sizeof(IntT));

  for (IntT edge_idx = 0; edge_idx < num_edges; edge_idx++) {
    fscanf(edgeFile, "%lu", srcs + edge_idx); // read src
    fscanf(edgeFile, "%lu", dsts + edge_idx); // read dst
    for (IntT feat_idx = 0; feat_idx < edge_feat_dim; feat_idx ++) {
        fscanf(edgeFile, "%lf", (FloatT *) (edge_feats + (edge_feat_dim * edge_idx) + feat_idx));
    }
  }

  // -------------------------
  // Build graph

  d_graph->num_nodes     = num_nodes;
  d_graph->num_edges     = num_edges;
  d_graph->node_feat_dim = node_feat_dim;
  d_graph->edge_feat_dim = edge_feat_dim;

  hipMalloc((void**)&d_graph->node_feats, num_nodes * node_feat_dim * sizeof(FloatT));
  hipMalloc((void**)&d_graph->edge_feats, num_edges * edge_feat_dim * sizeof(FloatT));
  hipMalloc((void**)&d_graph->srcs, num_edges * sizeof(IntT));
  hipMalloc((void**)&d_graph->dsts, num_edges * sizeof(IntT));

  hipMemcpy(d_graph->node_feats, node_feats, num_nodes * node_feat_dim * sizeof(FloatT), hipMemcpyHostToDevice);
  hipMemcpy(d_graph->edge_feats, edge_feats, num_edges * edge_feat_dim * sizeof(FloatT), hipMemcpyHostToDevice);
  hipMemcpy(d_graph->srcs, srcs, num_edges * sizeof(IntT), hipMemcpyHostToDevice);
  hipMemcpy(d_graph->dsts, dsts, num_edges * sizeof(IntT), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_graph->srcs_r, num_edges * sizeof(IntT));
  hipMalloc((void**)&d_graph->dsts_r, num_edges * sizeof(IntT));
  hipMalloc((void**)&d_graph->map_r, num_edges * sizeof(IntT));
}


int main ( int argc, char * argv[] ) {

  // --
  // IO


  Graph data;
  char* data_node_path = argv[1];
  char* data_edge_path = argv[2];
  loadGraph(data_node_path, data_edge_path, &data);

  Graph patt;
  char* patt_node_path = argv[3];
  char* patt_edge_path = argv[4];
  loadGraph(patt_node_path, patt_edge_path, &patt);

  // --
  // Allocate memory

  FloatT *CV,
         *CE,
         *MU,
         *RE,
         *FE,
         *VR,
         *VF,
         *VRmax,
         *VFmax,
         *RMax,
         *FMax;

  // FloatT *Cnull; // Ignoring for now

  hipMalloc((void **)&CV,    data.num_nodes * patt.num_nodes * sizeof(FloatT));
  hipMalloc((void **)&MU,    data.num_nodes * patt.num_nodes * sizeof(FloatT));

  hipMalloc((void **)&CE,    data.num_edges * patt.num_edges * sizeof(FloatT));
  hipMalloc((void **)&RE,    data.num_edges * patt.num_edges * sizeof(FloatT));
  hipMalloc((void **)&FE,    data.num_edges * patt.num_edges * sizeof(FloatT));

  hipMalloc((void **)&VR,    data.num_nodes * patt.num_edges * sizeof(FloatT));
  hipMalloc((void **)&VF,    data.num_nodes * patt.num_edges * sizeof(FloatT));
  hipMalloc((void **)&VRmax,                  patt.num_edges * sizeof(FloatT));
  hipMalloc((void **)&VFmax,                  patt.num_edges * sizeof(FloatT));
  hipMalloc((void **)&RMax,  data.num_nodes * patt.num_edges * sizeof(FloatT));
  hipMalloc((void **)&FMax,  data.num_nodes * patt.num_edges * sizeof(FloatT));
  // hipMalloc((void **)&Cnull,                  patt.num_edges * sizeof(FloatT));

  IntT block_vv = 1 + (data.num_nodes * patt.num_nodes) / THREAD;
  IntT block_ee = 1 + (data.num_edges * patt.num_edges) / THREAD;
  IntT block_ve = 1 + (data.num_nodes * patt.num_edges) / THREAD;

  // --
  // Start timer

  cuda_timer_t timer;
  timer.start();

  // --
  // Initialize algorithm

  ac::host::SortEdges(data.srcs, data.dsts, data.srcs_r, data.dsts_r, data.map_r, data.num_edges);
  ac::host::SortEdges(patt.srcs, patt.dsts, patt.srcs_r, patt.dsts_r, patt.map_r, patt.num_edges);

  // Node-node distance matrix
  ac::device::NodePairwiseNorm<<<block_vv, THREAD>>>(
    data.num_nodes,
    patt.num_nodes,
    CV,
    MU,
    data.node_feats,
    patt.node_feats,
    data.node_feat_dim
  );


  // Edge-edge distance matrix
  ac::device::EdgePairwiseNorm<<<block_ee, THREAD>>>(
    data.num_edges,
    patt.num_edges,
    CE,
    RE,
    FE,
    data.edge_feats,
    patt.edge_feats,
    data.edge_feat_dim
  );

  // Normalize distance matrices (could all happen in parallel)
  cuda_timer_t timer_;
  timer_.start();

  ac::host::ColumnSoftmax(data.num_nodes, patt.num_nodes, CV);
  ac::host::ColumnSoftmax(data.num_nodes, patt.num_nodes, MU);
  ac::host::ColumnSoftmax(data.num_edges, patt.num_edges, CE);
  ac::host::ColumnSoftmax(data.num_edges, patt.num_edges, RE);
  ac::host::ColumnSoftmax(data.num_edges, patt.num_edges, FE);

  long long elapsed_ = timer_.stop();
  std::cerr << "elapsed_=" << elapsed_ << std::endl;

  // Repeat columns of MU by pattern edgelist
  ac::device::RepeatColumnsByPatternEdges<<<block_ve, THREAD>>>(
    data.num_nodes,
    patt.num_edges,
    patt.num_nodes,
    MU,
    VR,
    VF,
    patt.srcs,
    patt.dsts
  );

  // Hardcode Cnull to 0
  // hipMemset(Cnull, 0, patt.num_edges * sizeof(FloatT));

  // Compute max over columns of VF/VR
  ac::host::ColumnMax(data.num_nodes, patt.num_edges, VF, VFmax);
  ac::host::ColumnMax(data.num_nodes, patt.num_edges, VR, VRmax);

  // Max reduce over edges adjacent to data nodes
  ac::host::EdgeMaxReduce(data.num_edges, data.num_nodes, patt.num_edges,
    VRmax, FE, FMax,
    data.dsts_r, data.map_r
  );

  ac::host::EdgeMaxReduce(
    data.num_edges, data.num_nodes, patt.num_edges,
    VFmax, RE, RMax,
    data.srcs, NULL
  );

  // --
  // Run

  for (IntT i = 0; i < patt.num_nodes; i++) {
    // Repeat columns of (MU - FMax) by pattern edgelist
    ac::device::RepeatColumnsByPatternEdgesSubtract<<<block_ve, THREAD>>>(
      data.num_nodes,
      patt.num_edges,
      patt.num_nodes,
      MU,
      VR,
      VF,
      FMax,
      RMax,
      patt.srcs,
      patt.dsts
    );

    // Compute max over columns of VF/VR
    ac::host::ColumnMax(data.num_nodes, patt.num_edges, VF, VFmax);
    ac::host::ColumnMax(data.num_nodes, patt.num_edges, VR, VRmax);

    // Repeat rows of VF/VR by data srcs
    ac::device::RepeatColumnsByDataEdges<<<block_ee, THREAD>>>(
      data.num_edges,
      patt.num_edges,
      CE,
      VR,
      VF,
      FE,
      RE,
      data.srcs
    );
    ac::host::ColumnSoftmax(data.num_edges, patt.num_edges, FE);
    ac::host::ColumnSoftmax(data.num_edges, patt.num_edges, RE);

    // Max aggregation over edges adjacent to data nodes
    ac::host::EdgeMaxReduce(data.num_edges, data.num_nodes, patt.num_edges,
      VRmax, FE, FMax,
      data.dsts_r, data.map_r
    );

    ac::host::EdgeMaxReduce(
      data.num_edges, data.num_nodes, patt.num_edges,
      VFmax, RE, RMax,
      data.srcs, NULL
    );

    // Replace columns of MU w/ sum over FMax/RMax of adjacent edges + subtract CV
    ac::host::ComputeMU(&patt, data.num_nodes, CV, FMax, RMax, MU);
    ac::host::ColumnSoftmax(data.num_nodes, patt.num_nodes, MU);
  }

  // --
  // Stop timer

  long long elapsed = timer.stop();
  std::cerr << "elapsed=" << elapsed << std::endl;

  // --
  // Copy results to host and print

  FloatT *h_MU = (FloatT *) malloc(data.num_nodes * patt.num_nodes * sizeof(FloatT));
  hipMemcpy(h_MU, MU, data.num_nodes * patt.num_nodes * sizeof(FloatT), hipMemcpyDeviceToHost);
  for (IntT i = 0; i < data.num_nodes * patt.num_nodes; i ++) printf("%e\n", h_MU[i]);
}

#endif