#ifndef __MAIN_CU
#define __MAIN_CU

#include <iostream>
#include "main.h"

// void loadGraph(char * edge_filename, char * node_filename, Graph* graph) {

//   // -------------------------
//   // Read nodes

//   uint64_t num_nodes, node_feat_dim;
//   FILE * nodeFile = fopen(node_filename, "r");
//   if (! nodeFile) {printf("Cannot open node file %s\n", node_filename); exit(1);}

//   fscanf(nodeFile, "%lu %lu", & num_nodes, & node_feat_dim);
//   uint64_t * node_table = (uint64_t *) malloc(num_nodes * node_feat_dim * sizeof(uint64_t));
//   for (uint64_t i = 0; i < num_nodes * node_feat_dim; i += node_feat_dim) {
//     fscanf(nodeFile, "%lu", node_table + i); // read node id
//     for (uint64_t j = 1; j < node_feat_dim; j ++) {
//         fscanf(nodeFile, "%lf", (double *) (node_table + i + j)); // read attribute
//     }
//   }

//   // -------------------------
//   // Read edges

//   uint64_t num_edges, edge_feat_dim;
//   FILE * edgeFile = fopen(edge_filename, "r");
//   if (! edgeFile) {printf("Cannot open file %s\n", edge_filename); exit(1);}

//   fscanf(edgeFile, "%lu %lu", & num_edges, & edge_feat_dim);
//   uint64_t * edge_table = (uint64_t *) malloc(num_edges * edge_feat_dim * sizeof(uint64_t));

//   uint64_t * srcs = (uint64_t *) malloc(num_edges * sizeof(uint64_t));
//   uint64_t * dsts = (uint64_t *) malloc(num_edges * sizeof(uint64_t));

//   for (uint64_t i = 0; i < num_edges * edge_feat_dim; i += edge_feat_dim) {
//     fscanf(edgeFile, "%lu", srcs + i); // read src
//     fscanf(edgeFile, "%lu", dsts + i); // read dst
//     for (uint64_t j = 0; j < edge_feat_dim - 2; j ++) {
//         fscanf(edgeFile, "%lf", (double *) (edge_table + i + j));
//     }
//   }
// }

Graph constructGraph(Table * Vtable, Table * Etable) {
  Graph graph;
  graph.Vtable       = * Vtable;
  graph.Etable       = * Etable;
  graph.num_edges    = Etable->num_rows;
  graph.num_vertices = Vtable->num_rows;
  return graph;
}

Table readEdgeTable(char * edge_filename) {
  uint64_t num_edges, edge_feat_dim;
  FILE * tableFile = fopen(edge_filename, "r");
  if (! tableFile) {printf("Cannot open file %s\n", edge_filename); exit(1);}

  fscanf(tableFile, "%lu %lu", & num_edges, & edge_feat_dim);
  uint64_t * table = (uint64_t *) malloc(num_edges * edge_feat_dim * sizeof(uint64_t));

  for (uint64_t i = 0; i < num_edges * edge_feat_dim; i += edge_feat_dim) {
      fscanf(tableFile, "%lu", table + i);                          // read src id
      fscanf(tableFile, "%lu", table + i + 1);                      // read dst id

      for (uint64_t j = 2; j < edge_feat_dim; j ++) {
          fscanf(tableFile, "%lf", (double *) (table + i + j));     // read attribute
  }   }

  Table edgeTable;
  edgeTable.num_rows = num_edges;
  edgeTable.num_cols = edge_feat_dim;
  edgeTable.table    = table;
  return edgeTable;
}

Table readVertexTable(char * node_filename) {
  uint64_t num_nodes, node_feat_dim;
  FILE * tableFile = fopen(node_filename, "r");
  if (! tableFile) {printf("Cannot open file %s\n", node_filename); exit(1);}

  fscanf(tableFile, "%lu %lu", & num_nodes, & node_feat_dim);
  uint64_t * table = (uint64_t *) malloc(num_nodes * node_feat_dim * sizeof(uint64_t));

  for (uint64_t i = 0; i < num_nodes * node_feat_dim; i += node_feat_dim) {
      fscanf(tableFile, "%lu", table + i);                          // read id

      for (uint64_t j = 1; j < node_feat_dim; j ++) {
          fscanf(tableFile, "%lf", (double *) (table + i + j));     // read attribute
  }   }

  Table vertexTable;
  vertexTable.num_rows = num_nodes;
  vertexTable.num_cols = node_feat_dim;
  vertexTable.table    = table;
  return vertexTable;
}


void table2device(Table* d_table, Table* h_table) {
  d_table->num_rows = h_table->num_rows;
  d_table->num_cols = h_table->num_cols;

  hipMalloc((void**)&d_table->table, h_table->num_rows * h_table->num_cols * sizeof(uint64_t));
  hipMemcpy(d_table->table, h_table->table,
    h_table->num_rows * h_table->num_cols * sizeof(uint64_t), hipMemcpyHostToDevice);

  uint64_t h_srcs[d_table->num_rows];
  uint64_t h_dsts[d_table->num_rows];
  for(uint64_t i = 0; i < d_table->num_rows; i++) {
    h_srcs[i] = h_table->table[i * d_table->num_cols];
    h_dsts[i] = h_table->table[i * d_table->num_cols + 1];
  }

  hipMalloc((void**)&d_table->srcs, h_table->num_rows * sizeof(uint64_t));
  hipMalloc((void**)&d_table->dsts, h_table->num_rows * sizeof(uint64_t));
  hipMemcpy(d_table->srcs, h_srcs, h_table->num_rows * sizeof(uint64_t), hipMemcpyHostToDevice);
  hipMemcpy(d_table->dsts, h_dsts, h_table->num_rows * sizeof(uint64_t), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_table->srcs_r, h_table->num_rows * sizeof(uint64_t));
  hipMalloc((void**)&d_table->dsts_r, h_table->num_rows * sizeof(uint64_t));
}


int main ( int argc, char * argv[] ) {

  // --
  // IO

  Table h_Data_Vtable    = readVertexTable(argv[2]);
  Table h_Data_Etable    = readEdgeTable(argv[3]);
  Table h_Pattern_Vtable = readVertexTable(argv[4]);
  Table h_Pattern_Etable = readEdgeTable(argv[5]);

  Table d_Data_Vtable;
  table2device(&d_Data_Vtable, &h_Data_Vtable);

  Table d_Data_Etable;
  table2device(&d_Data_Etable, &h_Data_Etable);

  Table d_Pattern_Vtable;
  table2device(&d_Pattern_Vtable, &h_Pattern_Vtable);

  Table d_Pattern_Etable;
  table2device(&d_Pattern_Etable, &h_Pattern_Etable);

  Graph d_Data_Graph    = constructGraph(&d_Data_Vtable, &d_Data_Etable);
  Graph d_Pattern_Graph = constructGraph(&d_Pattern_Vtable, &d_Pattern_Etable);

  // --
  // Init

  WorkArrays d_WA;
  initializeWorkArrays(&d_Data_Graph, &d_Pattern_Graph, d_WA);

  // --
  // Run

  const uint64_t DV = d_Data_Graph.num_vertices;
  const uint64_t PV = d_Pattern_Graph.num_vertices;

  for (uint64_t i = 0; i < PV; i++) {
      run_iteration(&d_Data_Graph, &d_Pattern_Graph, d_WA);
  }

  // --
  // Print results

  double *h_MU = (double *) malloc(DV * PV * sizeof(double));
  hipMemcpy(h_MU, d_WA.MU, DV * PV * sizeof(double), hipMemcpyDeviceToHost);

  for (uint64_t i = 0; i < DV * PV; i ++) {
    printf("%e\n", h_MU[i]);
  }
  return 0;
}

#endif