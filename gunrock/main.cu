#ifndef __MAIN_CU
#define __MAIN_CU

#include <iostream>
#include "main.h"

Graph constructGraph(Table * Vtable, Table * Etable) {
  Graph graph;
  graph.Vtable       = * Vtable;
  graph.Etable       = * Etable;
  graph.num_edges    = Etable->num_rows;
  graph.num_vertices = Vtable->num_rows;
  return graph;
}



void table2device(Table* d_table, Table* h_table) {
  d_table->num_rows = h_table->num_rows;
  d_table->num_cols = h_table->num_cols;
  hipMalloc((void**)&d_table->table, h_table->num_rows * h_table->num_cols * sizeof(uint64_t));
  hipMemcpy(d_table->table, h_table->table, h_table->num_rows * h_table->num_cols * sizeof(uint64_t), hipMemcpyHostToDevice);
}


int main ( int argc, char * argv[] ) {

  // --
  // IO

  Table h_Data_Vtable    = readVertexTable(argv[2]);
  Table h_Data_Etable    = readEdgeTable(argv[3]);
  Table h_Pattern_Vtable = readVertexTable(argv[4]);
  Table h_Pattern_Etable = readEdgeTable(argv[5]);

  Table d_Data_Vtable;
  table2device(&d_Data_Vtable, &h_Data_Vtable);

  Table d_Data_Etable;
  table2device(&d_Data_Etable, &h_Data_Etable);

  Table d_Pattern_Vtable;
  table2device(&d_Pattern_Vtable, &h_Pattern_Vtable);

  Table d_Pattern_Etable;
  table2device(&d_Pattern_Etable, &h_Pattern_Etable);


  Graph h_Data_Graph    = constructGraph(&h_Data_Vtable, &h_Data_Etable);
  Graph h_Pattern_Graph = constructGraph(&h_Pattern_Vtable, &h_Pattern_Etable);

  Graph d_Data_Graph    = constructGraph(&d_Data_Vtable, &d_Data_Etable);
  Graph d_Pattern_Graph = constructGraph(&d_Pattern_Vtable, &d_Pattern_Etable);

  // --
  // Init
  WorkArrays h_WA;
  WorkArrays d_WA;
  initializeWorkArrays(
    &h_Data_Graph, &h_Pattern_Graph,
    &d_Data_Graph, &d_Pattern_Graph,
    h_WA, d_WA
  );

  // --
  // Run

  uint64_t DV = h_Data_Graph.num_vertices;
  uint64_t PV = h_Pattern_Graph.num_vertices;

  for (uint64_t iter = 0; iter < PV; iter ++) {
      run_iteration(&h_Data_Graph, &h_Pattern_Graph, h_WA, d_WA);
  }

  // --
  // Print results

  for (uint64_t i = 0; i < DV * PV; i ++) {
    printf("%e\n", h_WA.MU[i]);
  }
  return 0;
}

#endif